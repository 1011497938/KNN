#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""  
#include <hip/hip_runtime.h>
#include"hip/device_functions.h"

#include <stdio.h> 
#include<iostream>
#include <fstream>
#include <string>
#include "math.h"
#include<time.h>


//skiplist
#include <time.h>
#include <malloc.h>

#include <unordered_map>



using namespace std;
#define NEI_MAX	300	//�����������ƾ����С
#define PEOPLE_CAP 2500  //������
#define MOVIE_CAP  2500 //�ܵ�Ӱ��Ŀ
#define TEST_DATA_CAP 30000 //�ɷ����ĵ�Ӱ�����
#define THREDS_NUM 1024 //�������߳���

double approximate(double a)
{
	if (a == 0)
		return 1;

	if (a > 5)
		return 5;

	//if (a < 2.5)
	//	return 1;

//	return a;
	int b = (int)a;

	if (a - (double)b > 0.6)
		return (double)+ 1;
	if (a - (double)b > 0.3)
		return (double)b + 0.5;

	return (double)b;
}

//�������ƶ�
__global__ void getSim(const int testData[], const double rating_map[], double sim[], const int USER_NUM, const int MOVIE_NUM, const int offeset)
{
	int bid = blockIdx.x;   
	int tid = threadIdx.x;  
	
	int movie1Id = bid + 1;
	int movie2Id = ((tid > MOVIE_NUM >> 2) ? MOVIE_NUM - tid : tid) + 1;

	double movie1Sum = 0;
	double movie2Sum = 0;

	int kernelNum = 0;

	double movie1[PEOPLE_CAP];
	double movie2[PEOPLE_CAP];
	for (int i = 1; i <= USER_NUM; i++)
		if (rating_map[movie1Id * PEOPLE_CAP + i] && rating_map[movie2Id * PEOPLE_CAP + i])
		{
			movie1Sum += rating_map[movie1Id * PEOPLE_CAP + i];
			movie2Sum += rating_map[movie2Id * PEOPLE_CAP + i];
			movie1[kernelNum] = rating_map[movie1Id * PEOPLE_CAP + i];
			movie2[kernelNum] = rating_map[movie2Id * PEOPLE_CAP + i];
			kernelNum++;
		}
	if (kernelNum)
	{
		double bar1 = movie1Sum / kernelNum;
		double bar2 = movie2Sum / kernelNum;

		double temp1 = 0;
		double temp2 = 0;
		double temp3 = 0;

		for (int i = 0; i < kernelNum; i++)
		{
			temp1 += (movie1[i] - bar1)*(movie2[i] - bar2);
			temp2 += (movie1[i] - bar1)*(movie1[i] - bar1);
			temp3 += (movie2[i] - bar2)*(movie2[i] - bar2);
		}
		if (!temp2 || !temp1)
		{
			//sim[movie1Id * MOVIE_NUM + movie2Id] = 0;
			//sim[movie2Id * MOVIE_NUM + movie1Id] = 0;
		}
		else
		{
			double result = temp1 / sqrt(temp2 * temp3);
			//sim[movie1Id * MOVIE_NUM + movie2Id] = result > 0 ? result : 0;
			//sim[movie2Id * MOVIE_NUM + movie1Id] = result > 0 ? result : 0;
			sim[movie1Id * MOVIE_NUM + movie2Id] = result;
			sim[movie2Id * MOVIE_NUM + movie1Id] = result;
			//���Ǳ�Ԥ���movie															 
			//sim[testMovieId * MOVIE_NUM + tid] = 10;
		}

	}
	//else
	//{
	//	//sim[movie1Id * MOVIE_NUM + movie2Id] = 0;
	//	//sim[movie2Id * MOVIE_NUM + movie1Id] = 0;
	//}


}
__global__ void conclude(double result[], const double rating_map[], double sim[], const int TEST_NUM, const int MOVIE_NUM, const int testData[])
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int index = bid * blockDim.x + tid;

	if (index < TEST_NUM)
	{
		int testMovieId = testData[index] & 0xFFFF;   //��Ҫ�����ĵ�Ӱid
		int userId = testData[index] >> 16;

		double a = 0;
		double b = 0;
		double t1, t2, t3, t4;
		t1 = t2 = t3 = t4 = 0;
		int length = 0;
		double rate;
		double similarity;
		for (int i = 0; i < MOVIE_NUM; i++)
			if ( (rate = rating_map[i * PEOPLE_CAP + userId]) && (similarity = sim[testMovieId * MOVIE_NUM + i]))
			{
				t1 += similarity * similarity;
				t2 += similarity;
				t3 += similarity * rate;
				t4 += rate;
				length++;
			}

		a = (t3*length - t2*t4) / (t1*length - t2*t2);
		b = (t1*t4 - t2*t3) / (t1*length - t2*t2);

		result[index] = (a + b) > 0 ? a + b : 1;

		//double sum1 = 0;
		//double sum2 = 0;
		//double rate;
		//double similarity;
		//int length = 0;

		//double MaxSim = -1;
		//int MaxIndex = 0;
		//for (int i = 0; i < MOVIE_NUM; i++)
		//	if ((rate = rating_map[i * PEOPLE_CAP + userId]) && (similarity = sim[testMovieId * MOVIE_NUM + i]) && similarity > 0.8)
		//	{
		//		if ( similarity > MaxSim && i != testMovieId)
		//		{
		//			MaxSim = similarity;
		//			MaxIndex = i;
		//		}
		//		sum1 += rate * similarity;
		//		sum2 += similarity;
		//		length++;
		//	}
		//for (int i = 0; i < MOVIE_NUM; i++)
		//	if ((rate = rating_map[i * PEOPLE_CAP + userId]) && (similarity = sim[testMovieId * MOVIE_NUM + i]) && similarity > 0.7)
		//	{
		//		length++;
		//		if (length == 30)
		//			break;
		//		if (similarity > MaxSim && i != testMovieId)
		//		{
		//			MaxSim = similarity;
		//			MaxIndex = i;
		//		}
		//		sum1 += rate * similarity;
		//		sum2 += similarity;
		//	}
		//for (int i = 0; i < MOVIE_NUM; i++)
		//	if ((rate = rating_map[i * PEOPLE_CAP + userId]) && (similarity = sim[testMovieId * MOVIE_NUM + i]) && similarity > 0.6)
		//	{
		//		length++;
		//		if (length == 30)
		//			break;
		//		if (similarity > MaxSim && i != testMovieId)
		//		{
		//			MaxSim = similarity;
		//			MaxIndex = i;
		//		}
		//		sum1 += rate * similarity;
		//		sum2 += similarity;
		//	}

		//if (length < 3)
		//	result[index] = rating_map[MaxIndex * PEOPLE_CAP + userId];
		//else
		//	result[index] = sum1/sum2;
	}

}

double *dev_rating_map = 0;
double *temp_dev_rating_map = 0;
double *dev_sim = 0;
int *dev_test_data = 0;
double *dev_result = 0;


int main()
{
	//���Խ��
	int testData[TEST_DATA_CAP];
	double testRating[TEST_DATA_CAP];

	int startClock = clock();

	hipSetDevice(0);

	//�����ļ�
	int ReadingClock = clock();
	string fileName = "data/u1.base";
	ifstream ratingFile(fileName);
	if (!ratingFile.is_open())
	{
		std::cout << "Error opening " + fileName;
		exit(1);
	}

	//�����ļ�
	fileName = "data/u1.test";
	ifstream testFile(fileName);
	if (!testFile.is_open())
	{
		std::cout << "Error opening " + fileName;
		exit(1);
	}

	//����ļ�
	fileName = "data/result.txt";
	ofstream resultFile;
	resultFile.open(fileName);
	if (!resultFile.is_open())
	{
		std::cout << "Error opening " + fileName;
		exit(1);
	}

	//��������ļ�
	double rating;
	int userId, movieId;
	long long int timeStamp;

	int TEST_NUM = 0;
	while (!testFile.eof())
	{
		testFile >> userId >> movieId >> rating >> timeStamp;
		//int temp = userId << 16 | movieId;
		//cout <<  (temp >> 16) << " " << (temp & 0xFFFF) << endl;
		testRating[TEST_NUM] = rating;
		testData[TEST_NUM++] = userId << 16 | movieId;   //����idֵ��������65 536
	}
	hipMalloc((void**)&dev_test_data, TEST_NUM * sizeof(int));
	hipMemcpy(dev_test_data, testData, TEST_NUM * sizeof(int), hipMemcpyHostToDevice);
	testFile.close();

	//������������
	hipMalloc((void**)&dev_rating_map, PEOPLE_CAP * MOVIE_CAP * sizeof(double));   //���ǵ�Ӱ,������

	int MOVIE_NUM = 0;
	int PEOPLE_NUM = 0;
	while (!ratingFile.eof())
	{
		ratingFile >> userId >> movieId >> rating >> timeStamp;
		if (PEOPLE_NUM < userId)
			PEOPLE_NUM = userId;
		if (movieId > MOVIE_NUM)
			MOVIE_NUM = movieId;
		hipMemcpy(dev_rating_map + userId + movieId * PEOPLE_CAP, &rating, sizeof(double), hipMemcpyHostToDevice);
	}
	ratingFile.close();

	//double temp[20000];
	//for (int i = 0; i <  MOVIE_CAP; i++)
	//{
	//	cout << i << endl;
	//	hipMemcpy(temp, dev_rating_map + PEOPLE_CAP * i, PEOPLE_CAP * sizeof(double), hipMemcpyDeviceToHost);
	//	for (int j = 0; j < PEOPLE_CAP; j++)
	//		if (temp[j] != 0)
	//			cout << temp[j] << " ";
	//	cout << endl;
	//}

	cout << "There are " << MOVIE_NUM << " movies amd " << PEOPLE_NUM << " peoples" << endl;
	cout << TEST_NUM << " data need to be predicted" << endl;
	std::cout << "ReadFile use " << clock() - ReadingClock << "ms" << endl;


	hipMalloc((void**)&dev_sim, MOVIE_NUM * MOVIE_NUM * sizeof(double));   //simֵ
	int threadNum = MOVIE_NUM > 1024 ? 1024 : MOVIE_NUM;

	getSim <<< MOVIE_NUM, MOVIE_NUM /2>>> (dev_test_data, dev_rating_map, dev_sim, PEOPLE_NUM, MOVIE_NUM, 0);
	hipDeviceSynchronize();

	double temp[20000];
	for (int i = 1; i <= MOVIE_NUM; i++)
	{
		hipMemcpy(temp, dev_sim + MOVIE_NUM * i, MOVIE_NUM * sizeof(double), hipMemcpyDeviceToHost);
		for (int j = 0; j < MOVIE_NUM; j++)
			//if (temp[j] != 0)
			cout << i << " " << j << " " << temp[j] << endl;
	}

	hipMalloc((void**)&dev_result, TEST_NUM * sizeof(double));   //simֵ
	conclude << < 20, 1000 >> > (dev_result, dev_rating_map, dev_sim, TEST_NUM, MOVIE_NUM, dev_test_data);

	double *result = new double[TEST_NUM];
	hipMemcpy(result, dev_result, TEST_NUM * sizeof(double), hipMemcpyDeviceToHost);


	double d = 0;
	int realNum = 0;
	for (int i = 0; i < TEST_NUM; i++)
	{
		double predict = approximate(result[i]);
		//cout << (testData[i] >> 16) << " "  << (testData[i] & 0xFFFF) << " " << testRating[i] << " " << predict << endl;
		if ( predict != 1)
		{
			//cout << (testData[i] >> 16) << " " << (testData[i] & 0xFFFF) << " " << testRating[i] << " " << predict << endl;
			double c = fabs(testRating[i] - predict);
			d += c;
			realNum++;
		}

	}
		
	cout << "MAEΪ" << d / realNum << endl;

	hipFree(dev_rating_map);
	hipFree(dev_test_data);
	hipFree(dev_result);
	hipFree(dev_sim);
	std::cout << "total use " << clock() - startClock << "ms" << endl;
	return 0;
}

